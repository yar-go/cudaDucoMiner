#include "hip/hip_runtime.h"
/*
SHA-1 in C
By Steve Reid <steve@edmweb.com>
100% Public Domain
--------------------------------
CudaDucoMiner by Khaluza Yaroslav
*/

/* #define LITTLE_ENDIAN * This should be #define'd already, if true. */
/* #define SHA1HANDSOFF * Copies data before messing with it. */
#define SHA1HANDSOFF
#include <stdio.h>
#include <string.h>
/* for uint32_t */
#include <stdint.h>
typedef struct
{
    uint32_t state[5];
    uint32_t count[2];
    unsigned char buffer[64];
} SHA1_CTX;
#define rol(value, bits) (((value) << (bits)) | ((value) >> (32 - (bits))))
#if BYTE_ORDER == LITTLE_ENDIAN
#define blk0(i) (block->l[i] = (rol(block->l[i],24)&0xFF00FF00) \
    |(rol(block->l[i],8)&0x00FF00FF))
#elif BYTE_ORDER == BIG_ENDIAN
#define blk0(i) block->l[i]
#else
#error "Endianness not defined!"
#endif
#define blk(i) (block->l[i&15] = rol(block->l[(i+13)&15]^block->l[(i+8)&15] \
    ^block->l[(i+2)&15]^block->l[i&15],1))
#define R0(v,w,x,y,z,i) z+=((w&(x^y))^y)+blk0(i)+0x5A827999+rol(v,5);w=rol(w,30);
#define R1(v,w,x,y,z,i) z+=((w&(x^y))^y)+blk(i)+0x5A827999+rol(v,5);w=rol(w,30);
#define R2(v,w,x,y,z,i) z+=(w^x^y)+blk(i)+0x6ED9EBA1+rol(v,5);w=rol(w,30);
#define R3(v,w,x,y,z,i) z+=(((w|x)&y)|(w&x))+blk(i)+0x8F1BBCDC+rol(v,5);w=rol(w,30);
#define R4(v,w,x,y,z,i) z+=(w^x^y)+blk(i)+0xCA62C1D6+rol(v,5);w=rol(w,30);
__device__ void SHA1Transform(uint32_t state[5], const unsigned char buffer[64])
{
    uint32_t a, b, c, d, e;

    typedef union
    {
        unsigned char c[64];
        uint32_t l[16];
    } CHAR64LONG16;

#ifdef SHA1HANDSOFF
    CHAR64LONG16 block[1];      /* use array to appear as a pointer */

    memcpy(block, buffer, 64);
#else
    /* The following had better never be used because it causes the
     * pointer-to-const buffer to be cast into a pointer to non-const.
     * And the result is written through.  I threw a "const" in, hoping
     * this will cause a diagnostic.
     */
    CHAR64LONG16* block = (const CHAR64LONG16*)buffer;
#endif
    /* Copy context->state[] to working vars */
    a = state[0];
    b = state[1];
    c = state[2];
    d = state[3];
    e = state[4];
    /* 4 rounds of 20 operations each. Loop unrolled. */
    R0(a, b, c, d, e, 0);
    R0(e, a, b, c, d, 1);
    R0(d, e, a, b, c, 2);
    R0(c, d, e, a, b, 3);
    R0(b, c, d, e, a, 4);
    R0(a, b, c, d, e, 5);
    R0(e, a, b, c, d, 6);
    R0(d, e, a, b, c, 7);
    R0(c, d, e, a, b, 8);
    R0(b, c, d, e, a, 9);
    R0(a, b, c, d, e, 10);
    R0(e, a, b, c, d, 11);
    R0(d, e, a, b, c, 12);
    R0(c, d, e, a, b, 13);
    R0(b, c, d, e, a, 14);
    R0(a, b, c, d, e, 15);
    R1(e, a, b, c, d, 16);
    R1(d, e, a, b, c, 17);
    R1(c, d, e, a, b, 18);
    R1(b, c, d, e, a, 19);
    R2(a, b, c, d, e, 20);
    R2(e, a, b, c, d, 21);
    R2(d, e, a, b, c, 22);
    R2(c, d, e, a, b, 23);
    R2(b, c, d, e, a, 24);
    R2(a, b, c, d, e, 25);
    R2(e, a, b, c, d, 26);
    R2(d, e, a, b, c, 27);
    R2(c, d, e, a, b, 28);
    R2(b, c, d, e, a, 29);
    R2(a, b, c, d, e, 30);
    R2(e, a, b, c, d, 31);
    R2(d, e, a, b, c, 32);
    R2(c, d, e, a, b, 33);
    R2(b, c, d, e, a, 34);
    R2(a, b, c, d, e, 35);
    R2(e, a, b, c, d, 36);
    R2(d, e, a, b, c, 37);
    R2(c, d, e, a, b, 38);
    R2(b, c, d, e, a, 39);
    R3(a, b, c, d, e, 40);
    R3(e, a, b, c, d, 41);
    R3(d, e, a, b, c, 42);
    R3(c, d, e, a, b, 43);
    R3(b, c, d, e, a, 44);
    R3(a, b, c, d, e, 45);
    R3(e, a, b, c, d, 46);
    R3(d, e, a, b, c, 47);
    R3(c, d, e, a, b, 48);
    R3(b, c, d, e, a, 49);
    R3(a, b, c, d, e, 50);
    R3(e, a, b, c, d, 51);
    R3(d, e, a, b, c, 52);
    R3(c, d, e, a, b, 53);
    R3(b, c, d, e, a, 54);
    R3(a, b, c, d, e, 55);
    R3(e, a, b, c, d, 56);
    R3(d, e, a, b, c, 57);
    R3(c, d, e, a, b, 58);
    R3(b, c, d, e, a, 59);
    R4(a, b, c, d, e, 60);
    R4(e, a, b, c, d, 61);
    R4(d, e, a, b, c, 62);
    R4(c, d, e, a, b, 63);
    R4(b, c, d, e, a, 64);
    R4(a, b, c, d, e, 65);
    R4(e, a, b, c, d, 66);
    R4(d, e, a, b, c, 67);
    R4(c, d, e, a, b, 68);
    R4(b, c, d, e, a, 69);
    R4(a, b, c, d, e, 70);
    R4(e, a, b, c, d, 71);
    R4(d, e, a, b, c, 72);
    R4(c, d, e, a, b, 73);
    R4(b, c, d, e, a, 74);
    R4(a, b, c, d, e, 75);
    R4(e, a, b, c, d, 76);
    R4(d, e, a, b, c, 77);
    R4(c, d, e, a, b, 78);
    R4(b, c, d, e, a, 79);
    /* Add the working vars back into context.state[] */
    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
    state[4] += e;
    /* Wipe variables */
    a = b = c = d = e = 0;
#ifdef SHA1HANDSOFF
    memset(block, '\0', sizeof(block));
#endif
}
__device__ void SHA1Init(SHA1_CTX* context) {
    /* SHA1 initialization constants */
    context->state[0] = 0x67452301;
    context->state[1] = 0xEFCDAB89;
    context->state[2] = 0x98BADCFE;
    context->state[3] = 0x10325476;
    context->state[4] = 0xC3D2E1F0;
    context->count[0] = context->count[1] = 0;
}
__device__ void SHA1Update(SHA1_CTX* context, const unsigned char* data, uint32_t len) {
    uint32_t i;
    uint32_t j;
    j = context->count[0];
    if ((context->count[0] += len << 3) < j)
        context->count[1]++;
    context->count[1] += (len >> 29);
    j = (j >> 3) & 63;
    if ((j + len) > 63)
    {
        memcpy(&context->buffer[j], data, (i = 64 - j));
        SHA1Transform(context->state, context->buffer);
        for (; i + 63 < len; i += 64)
        {
            SHA1Transform(context->state, &data[i]);
        }
        j = 0;
    }
    else
        i = 0;
    memcpy(&context->buffer[j], &data[i], len - i);
}
__device__ void SHA1Final(unsigned char digest[20], SHA1_CTX* context)
{
    unsigned i;

    unsigned char finalcount[8];

    unsigned char c;

#if 0    /* untested "improvement" by DHR */
    /* Convert context->count to a sequence of bytes
     * in finalcount.  Second element first, but
     * big-endian order within element.
     * But we do it all backwards.
     */
    unsigned char* fcp = &finalcount[8];

    for (i = 0; i < 2; i++)
    {
        uint32_t t = context->count[i];

        int j;

        for (j = 0; j < 4; t >>= 8, j++)
            *--fcp = (unsigned char)t
    }
#else
    for (i = 0; i < 8; i++)
    {
        finalcount[i] = (unsigned char)((context->count[(i >= 4 ? 0 : 1)] >> ((3 - (i & 3)) * 8)) & 255);      /* Endian independent */
    }
#endif
    c = 0200;
    SHA1Update(context, &c, 1);
    while ((context->count[0] & 504) != 448)
    {
        c = 0000;
        SHA1Update(context, &c, 1);
    }
    SHA1Update(context, finalcount, 8); /* Should cause a SHA1Transform() */
    for (i = 0; i < 20; i++)
    {
        digest[i] = (unsigned char)
            ((context->state[i >> 2] >> ((3 - (i & 3)) * 8)) & 255);
    }
    /* Wipe variables */
    memset(context, '\0', sizeof(*context));
    memset(&finalcount, '\0', sizeof(finalcount));
}
__device__ void SHA1(char* hash_out, const char* str, int len) {
    SHA1_CTX ctx;
    unsigned int ii;

    SHA1Init(&ctx);
    for (ii = 0; ii < len; ii += 1) {
        SHA1Update(&ctx, (const unsigned char*)str + ii, 1);
    }
    SHA1Final((unsigned char*)hash_out, &ctx);
}
__device__ __host__ int strlenght(const char* str) {
    short i = 0;
    while (str[i] != '\0') {
        i++;
    }
    return i;
}
__device__ void toHex(char* to, unsigned num) {
    char alphabet[] = "0123456789abcdef";
    to[0] = alphabet[num / 16];
    to[1] = alphabet[num % 16];
    return;
}
__device__ void reverse(char *s, int length)
{
    int c;
    char* begin, * end, temp;

    //length = string_length(s);
    begin = s;
    end = s;

    for (c = 0; c < length - 1; c++)
        end++;

    for (c = 0; c < length / 2; c++)
    {
        temp = *end;
        *end = *begin;
        *begin = temp;

        begin++;
        end--;
    }
}

__device__ char* itoa(int num, char* str, int base = 10)
{
    int i = 0;
    bool isNegative = false;
    if (num == 0)
    {   str[i++] = '0';
        str[i] = '\0';
        return str;}
    if (num < 0 && base == 10){
        isNegative = true;
        num = -num;}

    while (num != 0){
        int rem = num % base;
        str[i++] = (rem > 9) ? (rem - 10) + 'a' : rem + '0';
        num = num / base;
    }
    if (isNegative)
        str[i++] = '-';
    str[i] = '\0'; // Append string terminator
    reverse(str, i);
    return str;
}

__device__ char* my_strcat(char* destination, const char* source){
    char* ptr = destination + strlenght(destination);
    while (*source != '\0') {
        *ptr++ = *source++;
    }
    *ptr = '\0';
    return destination;}

__device__ void stradd(char* to, int number) {
    const int lenght_str = strlenght(to);
    char tmp[100];
    itoa(number, tmp);
    my_strcat(to, tmp);
}

__device__ int strcomp(const char* X, const char* Y)
{
    while (*X)
    {
        if (*X != *Y) break;
        X++;
        Y++;
    }
    return *(const unsigned char*)X - *(const unsigned char*)Y;
}

#include <iostream>
using namespace std;

__global__ void kernel(char* victim, char* perfect, int span, int* answer) {
    if (*answer != 0) return;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    char result[21];
    char tmp[41];
    char tmp_victim[200];
    char tmp_perfect[200];

    for (int i = 0; i < 200; i++) {
        tmp_victim[i] = victim[i];
        tmp_perfect[i] = perfect[i];
    }
    int backss = strlenght(tmp_victim);

    int block_size = span / 30;
    int block_start = ((span / 30) * bid);
    int start_i = block_start + (block_size / 1024) * tid;
    int end_i = start_i + (block_size / 1024)+1;

    //for (int i = (span/1024)*tid; i < (span/1024)*tid + (span / 1024)+1; i++) {
    for( int i = start_i; i< end_i; i++){
        
        stradd(tmp_victim, i);
        SHA1(result, tmp_victim, strlenght(tmp_victim));
        for (int offset = 0; offset < 20; offset++) {
            toHex(tmp + (2 * offset), (unsigned)result[offset] & 0xff);
        }
        tmp[40] = '\0';
        if (strcomp(tmp, tmp_perfect) == 0) {*answer = i; break;}
        else tmp_victim[backss] = '\0';
    }
}

#include <sstream>

int main(int argc, char* argv[]){
    //char victim[500];
    //char perfect[500];
    int span, *dev_answer, answer;
    //cin >> victim >> perfect >> span;
    if (argc < 2) {
        cout << "ERROR: so small count of elements ):";
    }

    char* victim = argv[1];
    char* perfect = argv[2];
    stringstream convert(argv[3]); 

    if (!(convert >> span)) {
        cout << "ERROR!!!";
        return;
    }
        

    span = span * 100 + 1;

    char* dev_victim, * dev_perfect;
    hipMalloc((void**)&dev_victim, sizeof(char) * 200);
    hipMalloc((void**)&dev_perfect, sizeof(char) * 200);
    hipMalloc((void**)&dev_answer, sizeof(int));

    hipMemcpy(dev_victim, victim, sizeof(char) * 200, hipMemcpyHostToDevice);
    hipMemcpy(dev_perfect, perfect, sizeof(char) * 200, hipMemcpyHostToDevice);

    kernel << <1000, 1024 >> > (dev_victim, dev_perfect, span, dev_answer);
    
    hipMemcpy(&answer, dev_answer, sizeof(int), hipMemcpyDeviceToHost);

    cout << answer;

}